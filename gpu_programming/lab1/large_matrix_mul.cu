#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Utility function declarations
void initializeMatrix(float *matrix, int rows, int cols);
void printMatrixSubset(float *matrix, int rows, int cols, const char *name);
void cpuMatrixMultiply(float *A, float *B, float *C, int m, int n, int k);
void compareResults(float *cpuResult, float *gpuResult, int size);

int main() {
    // Matrix dimensions
    int N = 1024;
    
    // Allocate host memory
    float *h_A = (float*)malloc(N * N * sizeof(float));
    float *h_B = (float*)malloc(N * N * sizeof(float));
    float *h_C_cpu = (float*)malloc(N * N * sizeof(float));
    float *h_C_gpu = (float*)malloc(N * N * sizeof(float));
    
    if (!h_A || !h_B || !h_C_cpu || !h_C_gpu) {
        fprintf(stderr, "Host memory allocation failed\n");
        return 1;
    }
    
    // Initialize matrices
    printf("Initializing matrices...\n");
    initializeMatrix(h_A, N, N);
    initializeMatrix(h_B, N, N);
    
    // Print small subsets to verify data
    printMatrixSubset(h_A, N, N, "Matrix A (subset)");
    printMatrixSubset(h_B, N, N, "Matrix B (subset)");
    
    // ---------------- CPU Matrix Multiplication ----------------
    printf("Performing CPU matrix multiplication...\n");
    
    clock_t cpu_start = clock();
    cpuMatrixMultiply(h_A, h_B, h_C_cpu, N, N, N);
    clock_t cpu_end = clock();
    
    double cpu_time = ((double)(cpu_end - cpu_start)) / CLOCKS_PER_SEC;
    printf("CPU matrix multiplication completed in %.3f seconds\n", cpu_time);
    
    // ---------------- cuBLAS Matrix Multiplication ----------------
    printf("Performing GPU matrix multiplication with cuBLAS...\n");
    
    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);
    
    // Perform matrix multiplication using cuBLAS
    float alpha = 1.0f;
    float beta = 0.0f;
    
    hipEventRecord(start);
    
    // Note: cuBLAS uses column-major order, so we compute B * A instead of A * B
    // C = alpha*op(A)*op(B) + beta*C
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                N, N, N, 
                &alpha, 
                d_B, N,  // Matrix B
                d_A, N,  // Matrix A
                &beta, 
                d_C, N); // Matrix C result
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Copy result from device to host
    hipMemcpy(h_C_gpu, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);
    
    // Calculate elapsed time
    float gpu_time = 0;
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("GPU matrix multiplication completed in %.3f seconds\n", gpu_time/1000.0);
    
    // Compare results
    printMatrixSubset(h_C_cpu, N, N, "CPU Result (subset)");
    printMatrixSubset(h_C_gpu, N, N, "GPU Result (subset)");
    compareResults(h_C_cpu, h_C_gpu, N * N);
    
    // Print performance comparison
    printf("\nPerformance Comparison:\n");
    printf("CPU time: %.3f seconds\n", cpu_time);
    printf("GPU time: %.3f seconds\n", gpu_time/1000.0);
    printf("Speedup: %.2fx\n", cpu_time/(gpu_time/1000.0));
    
    // Calculate GFLOPS (Giga Floating Point Operations Per Second)
    double cpu_gflops = (2.0 * N * N * N) / (cpu_time * 1e9);
    double gpu_gflops = (2.0 * N * N * N) / ((gpu_time/1000.0) * 1e9);
    printf("CPU Performance: %.2f GFLOPS\n", cpu_gflops);
    printf("GPU Performance: %.2f GFLOPS\n", gpu_gflops);
    
    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    printf("\n--- Complexity Analysis ---\n");
    printf("Native CPU Implementation: O(N³) complexity\n");
    printf("- Triple-nested loops iterating through all matrix elements\n");
    printf("- No optimization for cache locality or parallelism\n\n");
    
    printf("cuBLAS Implementation: Effectively O(N³) but highly optimized\n");
    printf("- Uses tiling to maximize cache utilization\n");
    printf("- Employs thousands of parallel threads on GPU\n");
    printf("- Utilizes specialized matrix multiplication hardware (Tensor Cores if available)\n");
    printf("- Implements advanced blocking strategies to minimize memory access latency\n");
    printf("- Benefits from decades of research in optimizing matrix operations\n");
    
    return 0;
}

// Initialize matrix with random values
void initializeMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            matrix[i * cols + j] = (float)(rand() % 100) / 100.0f;
        }
    }
}

// Print a small subset of the matrix to verify data
void printMatrixSubset(float *matrix, int rows, int cols, const char *name) {
    printf("%s (3x3 corner):\n", name);
    int display_size = 3;
    for (int i = 0; i < display_size && i < rows; i++) {
        for (int j = 0; j < display_size && j < cols; j++) {
            printf("%.4f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

// Native CPU matrix multiplication implementation
void cpuMatrixMultiply(float *A, float *B, float *C, int m, int n, int k) {
    // A: m x k matrix
    // B: k x n matrix
    // C: m x n matrix (result)
    
    // Classic triple loop matrix multiplication
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            float sum = 0.0f;
            for (int p = 0; p < k; p++) {
                sum += A[i * k + p] * B[p * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

// Compare CPU and GPU results for accuracy
void compareResults(float *cpuResult, float *gpuResult, int size) {
    float epsilon = 1e-3;  // Tolerance for floating point comparison
    int errors = 0;
    
    for (int i = 0; i < size; i++) {
        if (fabs(cpuResult[i] - gpuResult[i]) > epsilon) {
            errors++;
            if (errors < 10) {
                printf("Error at index %d: CPU = %f, GPU = %f\n", 
                      i, cpuResult[i], gpuResult[i]);
            }
        }
    }
    
    if (errors > 0) {
        printf("Found %d errors (tolerance: %e)\n", errors, epsilon);
    } else {
        printf("Results match! No errors found.\n");
    }
}
